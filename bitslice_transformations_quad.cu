
#include <hip/hip_runtime.h>
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ < 300
/**
 * __shfl() returns the value of var held by the thread whose ID is given by srcLane.
 * If srcLane is outside the range 0..width-1, the thread's own value of var is returned.
 */
#undef __shfl
#define __shfl(var, srcLane, width) (uint32_t)(var)
#endif

#ifdef __CUDA_ARCH__

__device__ __forceinline__
void to_bitslice_quad(uint32_t * __restrict__ input, uint32_t * __restrict__ output)
{
    uint32_t other[8];
    const int n = threadIdx.x % 4;

    #pragma unroll
    for (int i = 0; i < 8; i++) {
        input[i] = __shfl((int)input[i], n ^ (3*(n >=1 && n <=2)), 4);
        other[i] = __shfl((int)input[i], (threadIdx.x + 1) % 4, 4);
        input[i] = __shfl((int)input[i], threadIdx.x & 2, 4);
        other[i] = __shfl((int)other[i], threadIdx.x & 2, 4);
        if (threadIdx.x & 1) {
            input[i] = __byte_perm(input[i], 0, 0x1032);
            other[i] = __byte_perm(other[i], 0, 0x1032);
        }
    }

    output[ 0] =  (input[ 0] & 0x00000001);
    output[ 0] |= ((other[ 0] & 0x00000001) << 1);
    output[ 0] |= ((input[ 1] & 0x00000001) << 2);
    output[ 0] |= ((other[ 1] & 0x00000001) << 3);
    output[ 0] |= ((input[ 2] & 0x00000001) << 4);
    output[ 0] |= ((other[ 2] & 0x00000001) << 5);
    output[ 0] |= ((input[ 3] & 0x00000001) << 6);
    output[ 0] |= ((other[ 3] & 0x00000001) << 7);
    output[ 0] |= ((input[ 4] & 0x00000001) << 8);
    output[ 0] |= ((other[ 4] & 0x00000001) << 9);
    output[ 0] |= ((input[ 5] & 0x00000001) <<10);
    output[ 0] |= ((other[ 5] & 0x00000001) <<11);
    output[ 0] |= ((input[ 6] & 0x00000001) <<12);
    output[ 0] |= ((other[ 6] & 0x00000001) <<13);
    output[ 0] |= ((input[ 7] & 0x00000001) <<14);
    output[ 0] |= ((other[ 7] & 0x00000001) <<15);
    output[ 0] |= ((input[ 0] & 0x00000100) << 8);
    output[ 0] |= ((other[ 0] & 0x00000100) << 9);
    output[ 0] |= ((input[ 1] & 0x00000100) <<10);
    output[ 0] |= ((other[ 1] & 0x00000100) <<11);
    output[ 0] |= ((input[ 2] & 0x00000100) <<12);
    output[ 0] |= ((other[ 2] & 0x00000100) <<13);
    output[ 0] |= ((input[ 3] & 0x00000100) <<14);
    output[ 0] |= ((other[ 3] & 0x00000100) <<15);
    output[ 0] |= ((input[ 4] & 0x00000100) <<16);
    output[ 0] |= ((other[ 4] & 0x00000100) <<17);
    output[ 0] |= ((input[ 5] & 0x00000100) <<18);
    output[ 0] |= ((other[ 5] & 0x00000100) <<19);
    output[ 0] |= ((input[ 6] & 0x00000100) <<20);
    output[ 0] |= ((other[ 6] & 0x00000100) <<21);
    output[ 0] |= ((input[ 7] & 0x00000100) <<22);
    output[ 0] |= ((other[ 7] & 0x00000100) <<23);

    output[ 1] = ((input[ 0] & 0x00000002) >> 1);
    output[ 1] |=  (other[ 0] & 0x00000002);
    output[ 1] |= ((input[ 1] & 0x00000002) << 1);
    output[ 1] |= ((other[ 1] & 0x00000002) << 2);
    output[ 1] |= ((input[ 2] & 0x00000002) << 3);
    output[ 1] |= ((other[ 2] & 0x00000002) << 4);
    output[ 1] |= ((input[ 3] & 0x00000002) << 5);
    output[ 1] |= ((other[ 3] & 0x00000002) << 6);
    output[ 1] |= ((input[ 4] & 0x00000002) << 7);
    output[ 1] |= ((other[ 4] & 0x00000002) << 8);
    output[ 1] |= ((input[ 5] & 0x00000002) << 9);
    output[ 1] |= ((other[ 5] & 0x00000002) <<10);
    output[ 1] |= ((input[ 6] & 0x00000002) <<11);
    output[ 1] |= ((other[ 6] & 0x00000002) <<12);
    output[ 1] |= ((input[ 7] & 0x00000002) <<13);
    output[ 1] |= ((other[ 7] & 0x00000002) <<14);
    output[ 1] |= ((input[ 0] & 0x00000200) << 7);
    output[ 1] |= ((other[ 0] & 0x00000200) << 8);
    output[ 1] |= ((input[ 1] & 0x00000200) << 9);
    output[ 1] |= ((other[ 1] & 0x00000200) <<10);
    output[ 1] |= ((input[ 2] & 0x00000200) <<11);
    output[ 1] |= ((other[ 2] & 0x00000200) <<12);
    output[ 1] |= ((input[ 3] & 0x00000200) <<13);
    output[ 1] |= ((other[ 3] & 0x00000200) <<14);
    output[ 1] |= ((input[ 4] & 0x00000200) <<15);
    output[ 1] |= ((other[ 4] & 0x00000200) <<16);
    output[ 1] |= ((input[ 5] & 0x00000200) <<17);
    output[ 1] |= ((other[ 5] & 0x00000200) <<18);
    output[ 1] |= ((input[ 6] & 0x00000200) <<19);
    output[ 1] |= ((other[ 6] & 0x00000200) <<20);
    output[ 1] |= ((input[ 7] & 0x00000200) <<21);
    output[ 1] |= ((other[ 7] & 0x00000200) <<22);

    output[ 2] = ((input[ 0] & 0x00000004) >> 2);
    output[ 2] |= ((other[ 0] & 0x00000004) >> 1);
    output[ 2] |=  (input[ 1] & 0x00000004);
    output[ 2] |= ((other[ 1] & 0x00000004) << 1);
    output[ 2] |= ((input[ 2] & 0x00000004) << 2);
    output[ 2] |= ((other[ 2] & 0x00000004) << 3);
    output[ 2] |= ((input[ 3] & 0x00000004) << 4);
    output[ 2] |= ((other[ 3] & 0x00000004) << 5);
    output[ 2] |= ((input[ 4] & 0x00000004) << 6);
    output[ 2] |= ((other[ 4] & 0x00000004) << 7);
    output[ 2] |= ((input[ 5] & 0x00000004) << 8);
    output[ 2] |= ((other[ 5] & 0x00000004) << 9);
    output[ 2] |= ((input[ 6] & 0x00000004) <<10);
    output[ 2] |= ((other[ 6] & 0x00000004) <<11);
    output[ 2] |= ((input[ 7] & 0x00000004) <<12);
    output[ 2] |= ((other[ 7] & 0x00000004) <<13);
    output[ 2] |= ((input[ 0] & 0x00000400) << 6);
    output[ 2] |= ((other[ 0] & 0x00000400) << 7);
    output[ 2] |= ((input[ 1] & 0x00000400) << 8);
    output[ 2] |= ((other[ 1] & 0x00000400) << 9);
    output[ 2] |= ((input[ 2] & 0x00000400) <<10);
    output[ 2] |= ((other[ 2] & 0x00000400) <<11);
    output[ 2] |= ((input[ 3] & 0x00000400) <<12);
    output[ 2] |= ((other[ 3] & 0x00000400) <<13);
    output[ 2] |= ((input[ 4] & 0x00000400) <<14);
    output[ 2] |= ((other[ 4] & 0x00000400) <<15);
    output[ 2] |= ((input[ 5] & 0x00000400) <<16);
    output[ 2] |= ((other[ 5] & 0x00000400) <<17);
    output[ 2] |= ((input[ 6] & 0x00000400) <<18);
    output[ 2] |= ((other[ 6] & 0x00000400) <<19);
    output[ 2] |= ((input[ 7] & 0x00000400) <<20);
    output[ 2] |= ((other[ 7] & 0x00000400) <<21);

    output[ 3] = ((input[ 0] & 0x00000008) >> 3);
    output[ 3] |= ((other[ 0] & 0x00000008) >> 2);
    output[ 3] |= ((input[ 1] & 0x00000008) >> 1);
    output[ 3] |=  (other[ 1] & 0x00000008);
    output[ 3] |= ((input[ 2] & 0x00000008) << 1);
    output[ 3] |= ((other[ 2] & 0x00000008) << 2);
    output[ 3] |= ((input[ 3] & 0x00000008) << 3);
    output[ 3] |= ((other[ 3] & 0x00000008) << 4);
    output[ 3] |= ((input[ 4] & 0x00000008) << 5);
    output[ 3] |= ((other[ 4] & 0x00000008) << 6);
    output[ 3] |= ((input[ 5] & 0x00000008) << 7);
    output[ 3] |= ((other[ 5] & 0x00000008) << 8);
    output[ 3] |= ((input[ 6] & 0x00000008) << 9);
    output[ 3] |= ((other[ 6] & 0x00000008) <<10);
    output[ 3] |= ((input[ 7] & 0x00000008) <<11);
    output[ 3] |= ((other[ 7] & 0x00000008) <<12);
    output[ 3] |= ((input[ 0] & 0x00000800) << 5);
    output[ 3] |= ((other[ 0] & 0x00000800) << 6);
    output[ 3] |= ((input[ 1] & 0x00000800) << 7);
    output[ 3] |= ((other[ 1] & 0x00000800) << 8);
    output[ 3] |= ((input[ 2] & 0x00000800) << 9);
    output[ 3] |= ((other[ 2] & 0x00000800) <<10);
    output[ 3] |= ((input[ 3] & 0x00000800) <<11);
    output[ 3] |= ((other[ 3] & 0x00000800) <<12);
    output[ 3] |= ((input[ 4] & 0x00000800) <<13);
    output[ 3] |= ((other[ 4] & 0x00000800) <<14);
    output[ 3] |= ((input[ 5] & 0x00000800) <<15);
    output[ 3] |= ((other[ 5] & 0x00000800) <<16);
    output[ 3] |= ((input[ 6] & 0x00000800) <<17);
    output[ 3] |= ((other[ 6] & 0x00000800) <<18);
    output[ 3] |= ((input[ 7] & 0x00000800) <<19);
    output[ 3] |= ((other[ 7] & 0x00000800) <<20);

    output[ 4] = ((input[ 0] & 0x00000010) >> 4);
    output[ 4] |= ((other[ 0] & 0x00000010) >> 3);
    output[ 4] |= ((input[ 1] & 0x00000010) >> 2);
    output[ 4] |= ((other[ 1] & 0x00000010) >> 1);
    output[ 4] |=  (input[ 2] & 0x00000010);
    output[ 4] |= ((other[ 2] & 0x00000010) << 1);
    output[ 4] |= ((input[ 3] & 0x00000010) << 2);
    output[ 4] |= ((other[ 3] & 0x00000010) << 3);
    output[ 4] |= ((input[ 4] & 0x00000010) << 4);
    output[ 4] |= ((other[ 4] & 0x00000010) << 5);
    output[ 4] |= ((input[ 5] & 0x00000010) << 6);
    output[ 4] |= ((other[ 5] & 0x00000010) << 7);
    output[ 4] |= ((input[ 6] & 0x00000010) << 8);
    output[ 4] |= ((other[ 6] & 0x00000010) << 9);
    output[ 4] |= ((input[ 7] & 0x00000010) <<10);
    output[ 4] |= ((other[ 7] & 0x00000010) <<11);
    output[ 4] |= ((input[ 0] & 0x00001000) << 4);
    output[ 4] |= ((other[ 0] & 0x00001000) << 5);
    output[ 4] |= ((input[ 1] & 0x00001000) << 6);
    output[ 4] |= ((other[ 1] & 0x00001000) << 7);
    output[ 4] |= ((input[ 2] & 0x00001000) << 8);
    output[ 4] |= ((other[ 2] & 0x00001000) << 9);
    output[ 4] |= ((input[ 3] & 0x00001000) <<10);
    output[ 4] |= ((other[ 3] & 0x00001000) <<11);
    output[ 4] |= ((input[ 4] & 0x00001000) <<12);
    output[ 4] |= ((other[ 4] & 0x00001000) <<13);
    output[ 4] |= ((input[ 5] & 0x00001000) <<14);
    output[ 4] |= ((other[ 5] & 0x00001000) <<15);
    output[ 4] |= ((input[ 6] & 0x00001000) <<16);
    output[ 4] |= ((other[ 6] & 0x00001000) <<17);
    output[ 4] |= ((input[ 7] & 0x00001000) <<18);
    output[ 4] |= ((other[ 7] & 0x00001000) <<19);

    output[ 5] = ((input[ 0] & 0x00000020) >> 5);
    output[ 5] |= ((other[ 0] & 0x00000020) >> 4);
    output[ 5] |= ((input[ 1] & 0x00000020) >> 3);
    output[ 5] |= ((other[ 1] & 0x00000020) >> 2);
    output[ 5] |= ((input[ 2] & 0x00000020) >> 1);
    output[ 5] |=  (other[ 2] & 0x00000020);
    output[ 5] |= ((input[ 3] & 0x00000020) << 1);
    output[ 5] |= ((other[ 3] & 0x00000020) << 2);
    output[ 5] |= ((input[ 4] & 0x00000020) << 3);
    output[ 5] |= ((other[ 4] & 0x00000020) << 4);
    output[ 5] |= ((input[ 5] & 0x00000020) << 5);
    output[ 5] |= ((other[ 5] & 0x00000020) << 6);
    output[ 5] |= ((input[ 6] & 0x00000020) << 7);
    output[ 5] |= ((other[ 6] & 0x00000020) << 8);
    output[ 5] |= ((input[ 7] & 0x00000020) << 9);
    output[ 5] |= ((other[ 7] & 0x00000020) <<10);
    output[ 5] |= ((input[ 0] & 0x00002000) << 3);
    output[ 5] |= ((other[ 0] & 0x00002000) << 4);
    output[ 5] |= ((input[ 1] & 0x00002000) << 5);
    output[ 5] |= ((other[ 1] & 0x00002000) << 6);
    output[ 5] |= ((input[ 2] & 0x00002000) << 7);
    output[ 5] |= ((other[ 2] & 0x00002000) << 8);
    output[ 5] |= ((input[ 3] & 0x00002000) << 9);
    output[ 5] |= ((other[ 3] & 0x00002000) <<10);
    output[ 5] |= ((input[ 4] & 0x00002000) <<11);
    output[ 5] |= ((other[ 4] & 0x00002000) <<12);
    output[ 5] |= ((input[ 5] & 0x00002000) <<13);
    output[ 5] |= ((other[ 5] & 0x00002000) <<14);
    output[ 5] |= ((input[ 6] & 0x00002000) <<15);
    output[ 5] |= ((other[ 6] & 0x00002000) <<16);
    output[ 5] |= ((input[ 7] & 0x00002000) <<17);
    output[ 5] |= ((other[ 7] & 0x00002000) <<18);

    output[ 6] = ((input[ 0] & 0x00000040) >> 6);
    output[ 6] |= ((other[ 0] & 0x00000040) >> 5);
    output[ 6] |= ((input[ 1] & 0x00000040) >> 4);
    output[ 6] |= ((other[ 1] & 0x00000040) >> 3);
    output[ 6] |= ((input[ 2] & 0x00000040) >> 2);
    output[ 6] |= ((other[ 2] & 0x00000040) >> 1);
    output[ 6] |=  (input[ 3] & 0x00000040);
    output[ 6] |= ((other[ 3] & 0x00000040) << 1);
    output[ 6] |= ((input[ 4] & 0x00000040) << 2);
    output[ 6] |= ((other[ 4] & 0x00000040) << 3);
    output[ 6] |= ((input[ 5] & 0x00000040) << 4);
    output[ 6] |= ((other[ 5] & 0x00000040) << 5);
    output[ 6] |= ((input[ 6] & 0x00000040) << 6);
    output[ 6] |= ((other[ 6] & 0x00000040) << 7);
    output[ 6] |= ((input[ 7] & 0x00000040) << 8);
    output[ 6] |= ((other[ 7] & 0x00000040) << 9);
    output[ 6] |= ((input[ 0] & 0x00004000) << 2);
    output[ 6] |= ((other[ 0] & 0x00004000) << 3);
    output[ 6] |= ((input[ 1] & 0x00004000) << 4);
    output[ 6] |= ((other[ 1] & 0x00004000) << 5);
    output[ 6] |= ((input[ 2] & 0x00004000) << 6);
    output[ 6] |= ((other[ 2] & 0x00004000) << 7);
    output[ 6] |= ((input[ 3] & 0x00004000) << 8);
    output[ 6] |= ((other[ 3] & 0x00004000) << 9);
    output[ 6] |= ((input[ 4] & 0x00004000) <<10);
    output[ 6] |= ((other[ 4] & 0x00004000) <<11);
    output[ 6] |= ((input[ 5] & 0x00004000) <<12);
    output[ 6] |= ((other[ 5] & 0x00004000) <<13);
    output[ 6] |= ((input[ 6] & 0x00004000) <<14);
    output[ 6] |= ((other[ 6] & 0x00004000) <<15);
    output[ 6] |= ((input[ 7] & 0x00004000) <<16);
    output[ 6] |= ((other[ 7] & 0x00004000) <<17);

    output[ 7] = ((input[ 0] & 0x00000080) >> 7);
    output[ 7] |= ((other[ 0] & 0x00000080) >> 6);
    output[ 7] |= ((input[ 1] & 0x00000080) >> 5);
    output[ 7] |= ((other[ 1] & 0x00000080) >> 4);
    output[ 7] |= ((input[ 2] & 0x00000080) >> 3);
    output[ 7] |= ((other[ 2] & 0x00000080) >> 2);
    output[ 7] |= ((input[ 3] & 0x00000080) >> 1);
    output[ 7] |=  (other[ 3] & 0x00000080);
    output[ 7] |= ((input[ 4] & 0x00000080) << 1);
    output[ 7] |= ((other[ 4] & 0x00000080) << 2);
    output[ 7] |= ((input[ 5] & 0x00000080) << 3);
    output[ 7] |= ((other[ 5] & 0x00000080) << 4);
    output[ 7] |= ((input[ 6] & 0x00000080) << 5);
    output[ 7] |= ((other[ 6] & 0x00000080) << 6);
    output[ 7] |= ((input[ 7] & 0x00000080) << 7);
    output[ 7] |= ((other[ 7] & 0x00000080) << 8);
    output[ 7] |= ((input[ 0] & 0x00008000) << 1);
    output[ 7] |= ((other[ 0] & 0x00008000) << 2);
    output[ 7] |= ((input[ 1] & 0x00008000) << 3);
    output[ 7] |= ((other[ 1] & 0x00008000) << 4);
    output[ 7] |= ((input[ 2] & 0x00008000) << 5);
    output[ 7] |= ((other[ 2] & 0x00008000) << 6);
    output[ 7] |= ((input[ 3] & 0x00008000) << 7);
    output[ 7] |= ((other[ 3] & 0x00008000) << 8);
    output[ 7] |= ((input[ 4] & 0x00008000) << 9);
    output[ 7] |= ((other[ 4] & 0x00008000) <<10);
    output[ 7] |= ((input[ 5] & 0x00008000) <<11);
    output[ 7] |= ((other[ 5] & 0x00008000) <<12);
    output[ 7] |= ((input[ 6] & 0x00008000) <<13);
    output[ 7] |= ((other[ 6] & 0x00008000) <<14);
    output[ 7] |= ((input[ 7] & 0x00008000) <<15);
    output[ 7] |= ((other[ 7] & 0x00008000) <<16);
}

__device__ __forceinline__
void from_bitslice_quad(uint32_t * __restrict__ input, uint32_t * __restrict__ output)
{
    output[ 0] = ((input[ 0] & 0x00000100) >> 8);
    output[ 0] |= ((input[ 1] & 0x00000100) >> 7);
    output[ 0] |= ((input[ 2] & 0x00000100) >> 6);
    output[ 0] |= ((input[ 3] & 0x00000100) >> 5);
    output[ 0] |= ((input[ 4] & 0x00000100) >> 4);
    output[ 0] |= ((input[ 5] & 0x00000100) >> 3);
    output[ 0] |= ((input[ 6] & 0x00000100) >> 2);
    output[ 0] |= ((input[ 7] & 0x00000100) >> 1);
    output[ 0] |= ((input[ 0] & 0x01000000) >>16);
    output[ 0] |= ((input[ 1] & 0x01000000) >>15);
    output[ 0] |= ((input[ 2] & 0x01000000) >>14);
    output[ 0] |= ((input[ 3] & 0x01000000) >>13);
    output[ 0] |= ((input[ 4] & 0x01000000) >>12);
    output[ 0] |= ((input[ 5] & 0x01000000) >>11);
    output[ 0] |= ((input[ 6] & 0x01000000) >>10);
    output[ 0] |= ((input[ 7] & 0x01000000) >> 9);
    output[ 2] = ((input[ 0] & 0x00000200) >> 9);
    output[ 2] |= ((input[ 1] & 0x00000200) >> 8);
    output[ 2] |= ((input[ 2] & 0x00000200) >> 7);
    output[ 2] |= ((input[ 3] & 0x00000200) >> 6);
    output[ 2] |= ((input[ 4] & 0x00000200) >> 5);
    output[ 2] |= ((input[ 5] & 0x00000200) >> 4);
    output[ 2] |= ((input[ 6] & 0x00000200) >> 3);
    output[ 2] |= ((input[ 7] & 0x00000200) >> 2);
    output[ 2] |= ((input[ 0] & 0x02000000) >>17);
    output[ 2] |= ((input[ 1] & 0x02000000) >>16);
    output[ 2] |= ((input[ 2] & 0x02000000) >>15);
    output[ 2] |= ((input[ 3] & 0x02000000) >>14);
    output[ 2] |= ((input[ 4] & 0x02000000) >>13);
    output[ 2] |= ((input[ 5] & 0x02000000) >>12);
    output[ 2] |= ((input[ 6] & 0x02000000) >>11);
    output[ 2] |= ((input[ 7] & 0x02000000) >>10);
    output[ 4] = ((input[ 0] & 0x00000400) >>10);
    output[ 4] |= ((input[ 1] & 0x00000400) >> 9);
    output[ 4] |= ((input[ 2] & 0x00000400) >> 8);
    output[ 4] |= ((input[ 3] & 0x00000400) >> 7);
    output[ 4] |= ((input[ 4] & 0x00000400) >> 6);
    output[ 4] |= ((input[ 5] & 0x00000400) >> 5);
    output[ 4] |= ((input[ 6] & 0x00000400) >> 4);
    output[ 4] |= ((input[ 7] & 0x00000400) >> 3);
    output[ 4] |= ((input[ 0] & 0x04000000) >>18);
    output[ 4] |= ((input[ 1] & 0x04000000) >>17);
    output[ 4] |= ((input[ 2] & 0x04000000) >>16);
    output[ 4] |= ((input[ 3] & 0x04000000) >>15);
    output[ 4] |= ((input[ 4] & 0x04000000) >>14);
    output[ 4] |= ((input[ 5] & 0x04000000) >>13);
    output[ 4] |= ((input[ 6] & 0x04000000) >>12);
    output[ 4] |= ((input[ 7] & 0x04000000) >>11);
    output[ 6] = ((input[ 0] & 0x00000800) >>11);
    output[ 6] |= ((input[ 1] & 0x00000800) >>10);
    output[ 6] |= ((input[ 2] & 0x00000800) >> 9);
    output[ 6] |= ((input[ 3] & 0x00000800) >> 8);
    output[ 6] |= ((input[ 4] & 0x00000800) >> 7);
    output[ 6] |= ((input[ 5] & 0x00000800) >> 6);
    output[ 6] |= ((input[ 6] & 0x00000800) >> 5);
    output[ 6] |= ((input[ 7] & 0x00000800) >> 4);
    output[ 6] |= ((input[ 0] & 0x08000000) >>19);
    output[ 6] |= ((input[ 1] & 0x08000000) >>18);
    output[ 6] |= ((input[ 2] & 0x08000000) >>17);
    output[ 6] |= ((input[ 3] & 0x08000000) >>16);
    output[ 6] |= ((input[ 4] & 0x08000000) >>15);
    output[ 6] |= ((input[ 5] & 0x08000000) >>14);
    output[ 6] |= ((input[ 6] & 0x08000000) >>13);
    output[ 6] |= ((input[ 7] & 0x08000000) >>12);
    output[ 8] = ((input[ 0] & 0x00001000) >>12);
    output[ 8] |= ((input[ 1] & 0x00001000) >>11);
    output[ 8] |= ((input[ 2] & 0x00001000) >>10);
    output[ 8] |= ((input[ 3] & 0x00001000) >> 9);
    output[ 8] |= ((input[ 4] & 0x00001000) >> 8);
    output[ 8] |= ((input[ 5] & 0x00001000) >> 7);
    output[ 8] |= ((input[ 6] & 0x00001000) >> 6);
    output[ 8] |= ((input[ 7] & 0x00001000) >> 5);
    output[ 8] |= ((input[ 0] & 0x10000000) >>20);
    output[ 8] |= ((input[ 1] & 0x10000000) >>19);
    output[ 8] |= ((input[ 2] & 0x10000000) >>18);
    output[ 8] |= ((input[ 3] & 0x10000000) >>17);
    output[ 8] |= ((input[ 4] & 0x10000000) >>16);
    output[ 8] |= ((input[ 5] & 0x10000000) >>15);
    output[ 8] |= ((input[ 6] & 0x10000000) >>14);
    output[ 8] |= ((input[ 7] & 0x10000000) >>13);
    output[10] = ((input[ 0] & 0x00002000) >>13);
    output[10] |= ((input[ 1] & 0x00002000) >>12);
    output[10] |= ((input[ 2] & 0x00002000) >>11);
    output[10] |= ((input[ 3] & 0x00002000) >>10);
    output[10] |= ((input[ 4] & 0x00002000) >> 9);
    output[10] |= ((input[ 5] & 0x00002000) >> 8);
    output[10] |= ((input[ 6] & 0x00002000) >> 7);
    output[10] |= ((input[ 7] & 0x00002000) >> 6);
    output[10] |= ((input[ 0] & 0x20000000) >>21);
    output[10] |= ((input[ 1] & 0x20000000) >>20);
    output[10] |= ((input[ 2] & 0x20000000) >>19);
    output[10] |= ((input[ 3] & 0x20000000) >>18);
    output[10] |= ((input[ 4] & 0x20000000) >>17);
    output[10] |= ((input[ 5] & 0x20000000) >>16);
    output[10] |= ((input[ 6] & 0x20000000) >>15);
    output[10] |= ((input[ 7] & 0x20000000) >>14);
    output[12] = ((input[ 0] & 0x00004000) >>14);
    output[12] |= ((input[ 1] & 0x00004000) >>13);
    output[12] |= ((input[ 2] & 0x00004000) >>12);
    output[12] |= ((input[ 3] & 0x00004000) >>11);
    output[12] |= ((input[ 4] & 0x00004000) >>10);
    output[12] |= ((input[ 5] & 0x00004000) >> 9);
    output[12] |= ((input[ 6] & 0x00004000) >> 8);
    output[12] |= ((input[ 7] & 0x00004000) >> 7);
    output[12] |= ((input[ 0] & 0x40000000) >>22);
    output[12] |= ((input[ 1] & 0x40000000) >>21);
    output[12] |= ((input[ 2] & 0x40000000) >>20);
    output[12] |= ((input[ 3] & 0x40000000) >>19);
    output[12] |= ((input[ 4] & 0x40000000) >>18);
    output[12] |= ((input[ 5] & 0x40000000) >>17);
    output[12] |= ((input[ 6] & 0x40000000) >>16);
    output[12] |= ((input[ 7] & 0x40000000) >>15);
    output[14] = ((input[ 0] & 0x00008000) >>15);
    output[14] |= ((input[ 1] & 0x00008000) >>14);
    output[14] |= ((input[ 2] & 0x00008000) >>13);
    output[14] |= ((input[ 3] & 0x00008000) >>12);
    output[14] |= ((input[ 4] & 0x00008000) >>11);
    output[14] |= ((input[ 5] & 0x00008000) >>10);
    output[14] |= ((input[ 6] & 0x00008000) >> 9);
    output[14] |= ((input[ 7] & 0x00008000) >> 8);
    output[14] |= ((input[ 0] & 0x80000000) >>23);
    output[14] |= ((input[ 1] & 0x80000000) >>22);
    output[14] |= ((input[ 2] & 0x80000000) >>21);
    output[14] |= ((input[ 3] & 0x80000000) >>20);
    output[14] |= ((input[ 4] & 0x80000000) >>19);
    output[14] |= ((input[ 5] & 0x80000000) >>18);
    output[14] |= ((input[ 6] & 0x80000000) >>17);
    output[14] |= ((input[ 7] & 0x80000000) >>16);

#pragma unroll 8
    for (int i = 0; i < 16; i+=2) {
        if (threadIdx.x & 1) output[i] = __byte_perm(output[i], 0, 0x1032);
        output[i] = __byte_perm(output[i], __shfl((int)output[i], (threadIdx.x+1)&3, 4), 0x7610);
        output[i+1] = __shfl((int)output[i], (threadIdx.x+2)&3, 4);
        if (threadIdx.x & 3) output[i] = output[i+1] = 0;
    }
}

#else

/* host "fake" functions */
#define from_bitslice_quad(st, out)
#define to_bitslice_quad(in, msg) in[0] = (uint32_t) in[0];

#endif /* device only code */
